#include "hip/hip_runtime.h"
// /usr/local/cuda-10.0/bin/nvcc -I. main.cu
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <vector>
#include <memory>

#include "vec3.h"
#include "utils.h"

#include <float.h>  // FLT_MAX
#include <assert.h>

using std::cout;
using std::endl;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line){
    if (result) {
        std::cerr << "CUDA error = " <<static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" <<func << "' \n";
        //Make sure we call CUDA Device Resetbefore exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;

        if ((*world)->hit(cur_ray, 0.001, MAXFLOAT, rec)) {

            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(r, rec, attenuation,scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }

        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}


__global__ void render_init(int max_x, int max_y, hiprandState *rand_state){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    // printf("Dim: %i %i %i \n", blockDim.x, blockDim.y, blockDim.z);
    if ((i >= max_x) || ( j >=max_y))
        return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequnce number, no offset
    //                vvvvvvvvvvv   regresion on higher resoution 1000
    // hiprand_init(0, pixel_index, 0 , &rand_state[pixel_index]);  //
    hiprand_init(pixel_index, 0, 0 , &rand_state[pixel_index]);  // this works as expected
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns,
                       camera **cam, hitable **world, hiprandState *rand_state
    ){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j>=max_y))
        return;

    int pixel_index = j * max_x + i;

    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for (int s=0; s<ns; s++) {  // sampling

        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }

    fb[pixel_index] = col/float(ns);
}


__global__ void create_world(hitable** d_list,
                  hitable **d_world,  /* ** - set refernce pointer to d_world */
                  int* total_objects,
                  camera **d_camera,
                  int nx, int ny){
    if (threadIdx.x == 0 && blockIdx.x ==0) {

        // ground
        int obj_index = 0;
        d_list[obj_index++] = new sphere(vec3(0,-100.5, -1), 100, new lambertian(vec3(0.8,0.8,0.8)));
        d_list[obj_index++] = new sphere(vec3(1,0, -1), 0.5, new lambertian(vec3(0.8,0.3,0.3)));
        d_list[obj_index++] = new sphere(vec3(0,0, -1), 0.5, new dielectric(1.5));
        d_list[obj_index++] = new sphere(vec3(-1,0, -1), 0.5, new metal(vec3(0.5,0.5,0.5), 0.8));

        vec3 look_from(3,3,2);
        vec3 look_at(0,0,-1);
        *d_camera = new camera(look_from,
                               look_at,
                               vec3(0,1,0),
                               20, float(nx)/float(ny),
                               0.5, /* 2.0 big aperture */
                               (look_from-look_at).length()/* distance to focus */
            );
        // At this point we can only verify if number of objects added to d_list matches initial value
        assert (*total_objects == obj_index);
        *d_world = new hitable_list(d_list, obj_index);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, int* total_objects, camera **d_camera){
    for (int i = 0; i < *total_objects; ++i){
        delete *(d_list + i);
    }
    delete *d_world;
    delete *d_camera;

}


__global__ void render(vec3 *fb, int max_x, int max_y){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j>=max_y))
        return;
    int pixel_index = j * max_x + i;

    fb[pixel_index] = vec3(float(i) / max_x,
                           float(j) / max_y,
                           0.2);

}

int main (){
    int ns = 10; // number of samples
    int nx = 600;
    int ny = 300;
    int num_pixels = nx*ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    //allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    int tx = 8;
    int ty = 8;

    // We need to know upfron tTotal number to create/allocate
    int *total_objects;
    hipMallocManaged( &total_objects,  sizeof( int ) );
    *total_objects = 4;

    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, (*total_objects)*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    create_world<<<1,1>>>(d_list, d_world, total_objects, d_camera, nx, ny );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    printf ("# objects created %i\n", *total_objects);

    //create a d_rand_state object for every pixel in our main routine.
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));


    clock_t start, stop;
    start = clock();

    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx,ty);

    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render <<<blocks, threads>>>(fb,nx,ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());  // no longer neccessary on Pascal or later

    stop = clock();
    double timer_seconds = ((double)(stop-start)) / CLOCKS_PER_SEC;
    std::cerr << "elapsed time:" << timer_seconds << std::endl;

    save_png(fb, nx, ny);
    save_ppm(fb, nx, ny);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list, d_world, total_objects, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));


    checkCudaErrors(hipFree(fb));

    return 0;
}
